#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <third_party/cub/cub/block/block_reduce.cuh>
#include "blas.h"
#include "mlfe/device_context/cuda_context.h"
#include "mlfe/core/device.h"

namespace mlfe{ namespace math{

template<>
void gemm<float, HIPContext>(const bool trans_a,
                              const bool trans_b,
                              const int m,
                              const int n,
                              const int k,
                              const float alpha,
                              const float *a_ptr,
                              const int lda,
                              const float *b_ptr,
                              const int ldb,
                              const float beta,
                              float *c_ptr,
                              const int ldc,
                              HIPContext *context
                             )
{
    hipblasOperation_t cuTransA =
        !trans_a ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        !trans_b ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    if (hipblasSgemm(context->GetHandler(),
        cuTransB, cuTransA,
        n, m, k,
        &alpha, b_ptr, (!trans_b) ? n : k,
        a_ptr, (!trans_a) ? k : m,
        &beta, c_ptr, n) != HIPBLAS_STATUS_SUCCESS) {
        throw std::string("gemm<float, HIPContext> : hipblasSgemm failed.");
    }
}

template<>
void gemm<double, HIPContext>(const bool trans_a,
                               const bool trans_b,
                               const int m,
                               const int n,
                               const int k,
                               const double alpha,
                               const double *a_ptr,
                               const int lda,
                               const double *b_ptr,
                               const int ldb,
                               const double beta,
                               double *c_ptr,
                               const int ldc,
                               HIPContext *context
                              )
{
    hipblasOperation_t cuTransA =
        !trans_a ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        !trans_b ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    if (hipblasDgemm(context->GetHandler(),
        cuTransB, cuTransA,
        n, m, k,
        &alpha, b_ptr, (!trans_b) ? n : k,
        a_ptr, (!trans_a) ? k : m,
        &beta, c_ptr, n) != HIPBLAS_STATUS_SUCCESS) {
        throw std::string("gemm<float, HIPContext> : hipblasDgemm failed.");
    }
}

template <>
void gemv<float, HIPContext>(const bool trans_a,
                              const int m,
                              const int n,
                              const float alpha,
                              const float *a_ptr,
                              const int lda,
                              const float *b_ptr,
                              const float beta,
                              float *c_ptr,
                              const int ldc,
                              HIPContext *context
                              )
{
    hipblasOperation_t cuTransA = (!trans_a) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    if (hipblasSgemv(
        context->GetHandler(),
        cuTransA, n, m,
        &alpha, a_ptr,
        n, b_ptr, 1,
        &beta, c_ptr, 1) != HIPBLAS_STATUS_SUCCESS) {
        throw std::string("gemv<float, HIPContext> : hipblasSgemv failed.");
    }
}

template <>
void gemv<double, HIPContext>(const bool trans_a,
                               const int m,
                               const int n,
                               const double alpha,
                               const double *a_ptr,
                               const int lda,
                               const double *b_ptr,
                               const double beta,
                               double *c_ptr,
                               const int ldc,
                               HIPContext *context
                              )
{
    hipblasOperation_t cuTransA = (!trans_a) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    if (hipblasDgemv(
        context->GetHandler(),
        cuTransA, n, m,
        &alpha, a_ptr,
        n, b_ptr, 1,
        &beta, c_ptr, 1) != HIPBLAS_STATUS_SUCCESS) {
        throw std::string("gemv<float, HIPContext> : hipblasDgemv failed.");
    }
}

} // end namespace math
} // end namespace mlfe
