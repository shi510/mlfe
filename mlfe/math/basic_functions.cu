#include "hip/hip_runtime.h"
#include "basic_functions.h"
#include <third_party/cub/cub/block/block_reduce.cuh>
#include "mlfe/math/blas.h"
#include "mlfe/device_context/cuda_context.h"

namespace mlfe{
namespace math{

template <class T> __global__
void negative_kernel(const int size,
                     const T *x_ptr,
                     T *y_ptr){
    CUDA_1D_KERNEL_LOOP(n, size){
        y_ptr[n] = -x_ptr[n];
    }
}

template <>
void negative<float, HIPContext>(const int size,
                                  const float *x_ptr,
                                  float *y_ptr
                                 ){
    negative_kernel <<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x_ptr, y_ptr);
}

template <class T> __global__
void squared_difference_kernel(const int size,
                               const T *x1_ptr,
                               const T *x2_ptr,
                               T *y_ptr){
    CUDA_1D_KERNEL_LOOP(n, size){
        y_ptr[n] = std::pow(x1_ptr[n] - x2_ptr[n], 2);
    }
}

template <>
void squared_difference<float, HIPContext>(const int size,
                                            const float *x1_ptr,
                                            const float *x2_ptr,
                                            float *y_ptr){
    squared_difference_kernel<<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x1_ptr, x2_ptr, y_ptr);
}

template <>
void squared_difference<double, HIPContext>(const int size,
                                             const double *x1_ptr,
                                             const double *x2_ptr,
                                             double *y_ptr){
    squared_difference_kernel<<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x1_ptr, x2_ptr, y_ptr);
}

template <class DataType> __global__
void rowwise_max_kernel(const int rows,
                        const int cols,
                        const DataType *data, DataType *out
                       )
{
    typedef hipcub::BlockReduce<float, CUDA_CONTEXT_NUM_THREADS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    for(int rowIndex = blockIdx.x; rowIndex < rows; rowIndex += gridDim.x){
        DataType maxval = static_cast<DataType>(-FLT_MAX);
        for(int colIndex = threadIdx.x; colIndex < cols; colIndex += blockDim.x){
            maxval = max(data[rowIndex * cols + colIndex], maxval);
        }
        maxval = BlockReduce(temp_storage).Reduce(maxval, hipcub::Max());
        if(threadIdx.x == 0){
            out[rowIndex] = maxval;
        }
        __syncthreads();
    }
}

template <> void
rowwise_max<float, HIPContext>(const int m,
                                const int n,
                                const float *a_ptr,
                                float *b_ptr
                               )
{
    rowwise_max_kernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(n),
        CUDA_CONTEXT_NUM_THREADS>>>(m, n, a_ptr, b_ptr);
}

template <> void
rowwise_max<double, HIPContext>(const int m,
                                 const int n,
                                 const double *a_ptr,
                                 double *b_ptr
                                )
{
    rowwise_max_kernel<double><<<
        CUDA_CONTEXT_GET_BLOCKS(n),
        CUDA_CONTEXT_NUM_THREADS>>>(m, n, a_ptr, b_ptr);
}

template <class DT>
__global__ void rowwise_normalize_kernel(const int nthreads,
                                         const int D,
                                         const DT* scales,
                                         DT* out
                                        )
{
    CUDA_1D_KERNEL_LOOP(index, nthreads){
        int n = index / D;
        out[index] /= scales[n];
    }
}

template <> void
rowwise_normalize<float, HIPContext>(const int m,
                                      const int n,
                                      const float *scaler_ptr,
                                      float *norm_dest
                                     )
{
    rowwise_normalize_kernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(m * n),
        CUDA_CONTEXT_NUM_THREADS>>>(m * n, n, scaler_ptr, norm_dest);
}

template <> void
rowwise_normalize<double, HIPContext>(const int m,
                                       const int n,
                                       const double *scaler_ptr,
                                       double *norm_dest
                                      )
{
    rowwise_normalize_kernel<double><<<
        CUDA_CONTEXT_GET_BLOCKS(m * n),
        CUDA_CONTEXT_NUM_THREADS>>>(m * n, n, scaler_ptr, norm_dest);
}


template <class DataType> __global__
void exp_kernel(const int size,
                const DataType *x,
                DataType *y
               )
{
    CUDA_1D_KERNEL_LOOP(index, size){
        y[index] = std::exp(x[index]);
    }
}

template<>
void exp<float, HIPContext>(const int size,
                             const float *x_ptr,
                             float *y_ptr
                            )
{
    exp_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x_ptr, y_ptr);
}

template<>
void exp<double, HIPContext>(const int size,
                              const double *x_ptr,
                              double *y_ptr
                             )
{
    exp_kernel<double><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x_ptr, y_ptr);
}

template <class DataType> __global__
void axpy_kernel(const int size,
                 const DataType a,
                 const DataType *x,
                 DataType *y
                )
{
    CUDA_1D_KERNEL_LOOP(index, size){
        y[index] = a * x[index] + y[index];
    }
}

template<> void
axpy<float, HIPContext>(int size,
                         const float alpha,
                         const float *x_ptr,
                         float *y_ptr
                        )
{
    axpy_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, alpha, x_ptr, y_ptr);
}

template<> void
axpy<double, HIPContext>(int size,
                          const double alpha,
                          const double *x_ptr,
                          double *y_ptr
                         )
{
    axpy_kernel<double><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, alpha, x_ptr, y_ptr);
}

template <class DataType> __global__
void scale_kernel(const int size,
                  const DataType *x,
                  const DataType a,
                  DataType *y
                 )
{
    CUDA_1D_KERNEL_LOOP(index, size){
        y[index] = a * x[index];
    }
}

template <> void
scal<float, HIPContext>(const int size,
                         const float alpha,
                         const float *x_ptr,
                         float *y_ptr
                        )
{
    scale_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x_ptr, alpha, y_ptr);
}

template <> void
scal<double, HIPContext>(const int size,
                          const double alpha,
                          const double *x_ptr,
                          double *y_ptr
                         )
{
    scale_kernel<double><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x_ptr, alpha, y_ptr);
}

// This code refference from https://gist.github.com/yzhwang/5120437
template <int BLOCK_THREADS, typename DataType> __global__
void SumKernel(const int N,
               const DataType* X,
               DataType* Y
              )
{
    typedef hipcub::BlockReduce<DataType, BLOCK_THREADS> BlockReduce;

    __shared__ typename BlockReduce::TempStorage smem_storage;

    DataType data = 0;
    if(blockIdx.x * BLOCK_THREADS + threadIdx.x < N){
        data = X[blockIdx.x * BLOCK_THREADS + threadIdx.x];
    }

    DataType aggregate = BlockReduce(smem_storage).Sum(data);
    if(threadIdx.x == 0){
        atomicAdd(Y, aggregate);
    }
}

template <>
void sum<float, HIPContext>(const int size,
                             const float *x_ptr,
                             float *y_ptr
                            )
{
    SumKernel<CUDA_CONTEXT_NUM_THREADS, float><<<
        CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS >>>(size, x_ptr, y_ptr);
}

template <class DataType> __global__
void set_kernel(const int size,
                const DataType val,
                DataType *x
               )
{
    CUDA_1D_KERNEL_LOOP(index, size){
        x[index] = val;
    }
}

template<>
void set<float, HIPContext>(const int size,
                             const float val,
                             float *x_ptr
                            )
{
    set_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, val, x_ptr);
}

template<>
void set<double, HIPContext>(const int size,
                              const double val,
                              double *x_ptr
                             )
{
    set_kernel<double><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, val, x_ptr);
}

template <class DataType> __global__
void elementsize_mul_kernel(const int size,
                            const DataType *a,
                            const DataType *b,
                            DataType *c
                           )
{
    CUDA_1D_KERNEL_LOOP(n, size){
        c[n] = a[n] * b[n];
    }
}

template<>
void elementwise_mul<float, HIPContext>(const int size,
                                         const float *a,
                                         const float *b,
                                         float *c
                                        )
{
    elementsize_mul_kernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, a, b, c);
}

template <typename T>
__global__ void clip_min_max_kernel(const int size,
                                    T *data,
                                    T min,
                                    T max
                                   )
{
    CUDA_1D_KERNEL_LOOP(n, size){
        if(data[n] > max){
            data[n] = max;
        }
        else if(data[n] < min){
            data[n] = min;
        }
    }
}

template <>
void clip_min_max<float, HIPContext>(const int size,
                                      float *data,
                                      float min,
                                      float max
                                     )
{
    clip_min_max_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, data, min, max);
}

template <typename T>
__global__ void shift_a_b_kernel(const unsigned int size,
                                 T *numbers,
                                 const T a,
                                 const T b
                                )
{
    const T scale = b - a;
    CUDA_1D_KERNEL_LOOP(n, size){
        numbers[n] = numbers[n] * scale + a;
    }
}

template <>
void shift_a_b<float, HIPContext>(int size, float *x, float a, float b){
    shift_a_b_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x, a, b);
}

template <class T> __global__
void bernoulli_dist_kernel(const int size,
                           const T prob,
                           T *bernoulli
                          )
{
    CUDA_1D_KERNEL_LOOP(n, size){
        bernoulli[n] = bernoulli[n] >= prob ? T(1) : T(0);
    }
}

template <>
void bernoulli_distribution<float, HIPContext>(const int size,
                                                const float prob,
                                                float *bernoulli
                                               )
{
    hiprandGenerateUniform(cuda_context_v2::create()->get_curand_generator(), bernoulli, size);
    bernoulli_dist_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, prob, bernoulli);
}

#define DEFINE_BINARY_OP_KERNEL(OpName, Expr)      \
template <typename T>                              \
__global__ void OpName##_binary_op(const int size, \
                                   const T *a,     \
                                   const T *b,     \
                                   T *c){          \
    CUDA_1D_KERNEL_LOOP(n, size){                  \
        c[n] = a[n] Expr b[n];                     \
    }                                              \
}                                                  \
template <typename T> __global__                   \
void OpName##_val_binary_op(const int size,        \
                            const T val,           \
                            const T *a,            \
                            T *c){                 \
    CUDA_1D_KERNEL_LOOP(n, size){                  \
        c[n] = val Expr a[n];                      \
    }                                              \
}

DEFINE_BINARY_OP_KERNEL(Add, +)
DEFINE_BINARY_OP_KERNEL(Sub, -)
DEFINE_BINARY_OP_KERNEL(Mul, *)
DEFINE_BINARY_OP_KERNEL(Div, / )

#define DEFINE_CUDA_BINARY_OP(OpName)                  \
template <>                                            \
void OpName##Cuda<float>(const int size,               \
                         const float *a,               \
                         const float *b,               \
                         float *c                      \
                        )                              \
{                                                      \
    OpName##_binary_op<float><<<                       \
        CUDA_CONTEXT_GET_BLOCKS(size),                 \
        CUDA_CONTEXT_NUM_THREADS>>>(size, a, b, c);    \
}                                                      \
template <>                                            \
void OpName##ValCuda<float>(const int size,            \
                            const float val,           \
                            const float *a,            \
                            float *c)                  \
{                                                      \
    OpName##_val_binary_op<float><<<                   \
        CUDA_CONTEXT_GET_BLOCKS(size),                 \
        CUDA_CONTEXT_NUM_THREADS>>>(size, val, a, c);  \
}                                                      \
template <>                                            \
void OpName##Cuda<double>(                             \
                          const int size,              \
                          const double *a,             \
                          const double *b,             \
                          double *c                    \
                         )                             \
{                                                      \
    OpName##_binary_op<double><<<                      \
        CUDA_CONTEXT_GET_BLOCKS(size),                 \
        CUDA_CONTEXT_NUM_THREADS>>>(size, a, b, c);    \
}                                                      \
template <>                                            \
void OpName##ValCuda<double>(const int size,           \
                             const double val,         \
                             const double *a,          \
                             double *c                 \
                            )                          \
{                                                      \
    OpName##_val_binary_op<double><<<                  \
        CUDA_CONTEXT_GET_BLOCKS(size),                 \
        CUDA_CONTEXT_NUM_THREADS>>>(size, val, a, c);  \
}

DEFINE_CUDA_BINARY_OP(Add)
DEFINE_CUDA_BINARY_OP(Sub)
DEFINE_CUDA_BINARY_OP(Mul)
DEFINE_CUDA_BINARY_OP(Div)

} // end namespace math
} // end namespace mlfe
