#include "hip/hip_runtime.h"
#include "mlfe/operators_v2/impl/cuda/kernel/col2im.h"
#include "mlfe/operators_v2/utils.h"
#include "mlfe/device_context/cuda_context.h"
#include <third_party/cub/cub/block/block_reduce.cuh>

namespace mlfe{
namespace cuda_kernel{

template <typename T> __global__
void col2im_nhwc_kernel(
    const T* data_col,
    const int IC,
    const int IH,
    const int IW,
    const int OH,
    const int OW,
    const int ksize,
    const int stride,
    const int pad,
    T* data_im
    )
{
    const int COL_SIZE = ksize * ksize * IC;

    CUDA_1D_KERNEL_LOOP(c, COL_SIZE){
        int h_offset = c / IC / ksize;
        int w_offset = (c / IC) % ksize;
        int c_im = c % IC;
        for (int h = 0; h < OH; ++h) {
            for (int w = 0; w < OW; ++w) {
                int im_row = h_offset + h * stride - pad;
                int im_col = w_offset + w * stride - pad;
                if (im_row >= 0 && im_col >= 0 && im_row < IH && im_col < IW){
                    int col_index = (c * OH + h) * OW + w;
                    data_im[c_im + IC *(im_col + im_row*IW)] += data_col[col_index];
                }
                
            }
        }
    }
}

template <>
void col2im_nhwc<float>(
    const float* data_col,
    const int IC,
    const int IH,
    const int IW,
    const int OH,
    const int OW,
    const int ksize,
    const int stride,
    const int padding,
    float* data_im)
{
    col2im_nhwc_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(ksize * ksize * IC),
        CUDA_CONTEXT_NUM_THREADS>>>(data_col, IC, IH, IW, OH, OW, ksize, stride, padding, data_im);
}

} // namespace cuda_kernel
} // namespace mlfe
