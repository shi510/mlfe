#include "hip/hip_runtime.h"
#include "mlfe/optimizers_v2/impl/cuda/kernel/adam.h"
#include "mlfe/device_context/cuda_context.h"

namespace mlfe{
namespace cuda_kernel{

template <typename T>
__global__ void adam_kernel(const int size,
                            T *w,
                            const T *dw,
                            T *m_hist,
                            T *v_hist,
                            const T lr,
                            const T beta1,
                            const T beta2,
                            const T eps
                           )
{
    T correction = lr * sqrt(T(1) - beta2) / (T(1) - beta1);
    CUDA_1D_KERNEL_LOOP(n, size){
        T g = dw[n];
        T mh = beta1 * m_hist[n] + (T(1) - beta1) * g;
        T vh = beta2 * v_hist[n] + (T(1) - beta2) * g * g;
        m_hist[n] = mh;
        v_hist[n] = vh;
        w[n] -= correction * mh / (sqrt(vh) + eps);
    }
}

template <>
void adam<float>(const int size,
    float *w, const float *dw, float *m_hist, float *v_hist,
    const float lr, const float beta1, const float beta2, const float eps
    )
{
    adam_kernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(
            size, w, dw, m_hist, v_hist,
            lr, beta1, beta2, eps
            );
}

} // namespace cuda_kernel
} // namespace mlfe